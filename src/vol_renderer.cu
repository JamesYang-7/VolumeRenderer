#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <cuda_gl_interop.h>
#include <imgui.h>
#include <imgui_impl_glfw.h>
#include <imgui_impl_opengl3.h>
#include <tiny-cuda-nn/common_host.h>
#include <json/json.hpp>
#include "vol_renderer/camera.h"
#include "vol_renderer/ray.h"
#include "vol_renderer/bbox.h"
#include "vol_renderer/volume_data.h"
#include "vol_renderer/data_loader.h"
#include "vol_renderer/transfer_function.h"
#include "vol_renderer/timer.h"

using json = nlohmann::json;


__global__ void processTextureKernel(uchar4* frame_buffer, uint32_t width, uint32_t height) {
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    uint32_t idx = y * width + x;

    frame_buffer[idx].x = (uint32_t)((float)x / width * 255.0f);
    frame_buffer[idx].y = (uint32_t)((float)y / height * 255.0f);
    frame_buffer[idx].z = 127;
}

__host__ __device__ void composite(
    const glm::vec3& color,
    const float& alpha,
    const glm::vec3& bg_color,
    const float& bg_alpha,
    glm::vec3& out_color,
    float& out_alpha
)
{
    out_color = alpha * color + (1.0f - alpha) * bg_color;
    out_alpha = alpha + (1.0f - alpha) * bg_alpha;
}

__global__ void rayTracingKernel(
    uchar4* frame_buffer, uint32_t width,
    uint32_t height, const Camera* camera,
    const AABB* bbox,
    const VolumeData<float>* volume,
    TransferFunctionType tf_type,
    const float step_size = 0.01f
)
{
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    uint32_t idx = y * width + x;

    float u = ((float)x + 0.5f) / width;
    float v = ((float)y + 0.5f) / height;

    Ray ray = camera->generateRay(u, v);
    float tmin, tmax;
    
    glm::vec3 color(0.0f, 0.0f, 0.5f);
    float alpha = 0.2f;
    glm::vec3 bg_color(0.0f);
    float bg_alpha = 0.0f;
    glm::vec3 out_color(0.0f);
    float out_alpha = 0.0f;
    unsigned char r = 0, g = 0, b = 0;
    if (bbox->ray_intersect(&ray, &tmin, &tmax)) {
        float t = tmax;
        while (t > tmin) {
            glm::vec3 p = ray.at(t);
            glm::vec3 local_pos = bbox->getLocalPos(p);
            float val = volume->at(local_pos);
            glm::vec4 rgba = getColor(val, tf_type);
            color = glm::vec3(rgba.x, rgba.y, rgba.z);
            alpha = rgba.w;
            composite(color, alpha, bg_color, bg_alpha, out_color, out_alpha);
            bg_color = out_color;
            bg_alpha = out_alpha;
            t -= step_size;
        }
        r = (unsigned char)(255.0f * out_color.x);
        g = (unsigned char)(255.0f * out_color.y);
        b = (unsigned char)(255.0f * out_color.z);
    }
    frame_buffer[idx].x = r;
    frame_buffer[idx].y = g;
    frame_buffer[idx].z = b;
}

void rayTracingHost(
    uchar4* frame_buffer, uint32_t width,
    uint32_t height, const Camera* camera,
    const AABB* bbox,
    const VolumeData<float>* volume,
    TransferFunctionType tf_type,
    const float step_size = 0.01f
)
{
    for (uint32_t x = 0; x < width; ++x) {
        for (uint32_t y = 0; y < height; ++y) {
            uint32_t idx = y * width + x;
            float u = ((float)x + 0.5f) / width;
            float v = ((float)y + 0.5f) / height;

            Ray ray = camera->generateRay(u, v);
            float tmin, tmax;
            glm::vec3 color(0.0f, 0.0f, 0.5f);
            float alpha = 0.2f;
            glm::vec3 bg_color(0.0f);
            float bg_alpha = 0.0f;
            glm::vec3 out_color(0.0f);
            float out_alpha = 0.0f;
            unsigned char r = 0, g = 0, b = 0, a = 0;
            if (bbox->ray_intersect(&ray, &tmin, &tmax)) {
                float t = tmax;
                while (t > tmin) {
                    glm::vec3 p = ray.at(t);
                    glm::vec3 local_pos = bbox->getLocalPos(p);
                    float val = volume->at(local_pos);
                    glm::vec4 rgba = getColor(val, tf_type);
                    color = glm::vec3(rgba.x, rgba.y, rgba.z);
                    alpha = rgba.w;
                    composite(color, alpha, bg_color, bg_alpha, out_color, out_alpha);
                    bg_color = out_color;
                    bg_alpha = out_alpha;
                    t -= step_size;
                }
                r = (unsigned char)(255.0f * out_color.x);
                g = (unsigned char)(255.0f * out_color.y);
                b = (unsigned char)(255.0f * out_color.z);
                a = (unsigned char)(255.0f * out_alpha);
            }
            frame_buffer[idx].x = r;
            frame_buffer[idx].y = g;
            frame_buffer[idx].z = b;
            frame_buffer[idx].w = a;
        }
    }
}


struct VolumeRenderer {
    uint32_t res_x = 0;
    uint32_t res_y = 0;
    GLuint gl_texture_id;
    hipGraphicsResource* cuda_resource;
    uchar4* frame_buffer;
    size_t size;
    TransferFunctionType tf = TransferFunctionType::GRAYSCALE;
    float sampling_rate = 10.0f; // inverse of step size
    bool on_host = false;

    VolumeRenderer(uint32_t res_x, uint32_t res_y, bool on_host=false) : res_x(res_x), res_y(res_y), on_host(on_host) {
        size = res_x * res_y * sizeof(uchar4);
        if (on_host) {
            frame_buffer = (uchar4*)malloc(size);
            memset(frame_buffer, 255, size);
        } else {
            hipMalloc(&frame_buffer, size);
            hipMemset(frame_buffer, 255, size);
        }
    }

    ~VolumeRenderer() {
        if (on_host) {
            free(frame_buffer);
        } else {
            hipGraphicsUnregisterResource(cuda_resource);
            hipFree(frame_buffer);
        }
    }

    void init() {
        glGenTextures(1, &gl_texture_id);
        glBindTexture(GL_TEXTURE_2D, gl_texture_id);
        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, res_x, res_y, 0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
        if (!on_host) hipGraphicsGLRegisterImage(&cuda_resource, gl_texture_id, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
    }

    void renderTexture() {
        hipArray* array;
        hipGraphicsMapResources(1, &cuda_resource);
        hipGraphicsSubResourceGetMappedArray(&array, cuda_resource, 0, 0);

        dim3 blockSize(16, 16);
        dim3 numBlocks((res_x + blockSize.x - 1) / blockSize.x, 
                       (res_y + blockSize.y - 1) / blockSize.y);
        processTextureKernel<<<numBlocks, blockSize>>>(frame_buffer, res_x, res_y);

        hipMemcpyToArray(array, 0, 0, frame_buffer, size, hipMemcpyDeviceToDevice);
        hipDeviceSynchronize();
        hipGraphicsUnmapResources(1, &cuda_resource);
    }

    void render(const Camera& camera, const AABB& bbox, const VolumeData<float>* volume) {
        if (on_host) {
            render_host(camera, bbox, volume);
        } else {
            render_device(camera, bbox, volume);
        }
    }

    void render_device(const Camera& camera, const AABB& bbox, const VolumeData<float>* volume) {
        hipArray* array;
        hipGraphicsMapResources(1, &cuda_resource);
        hipGraphicsSubResourceGetMappedArray(&array, cuda_resource, 0, 0);

        dim3 blockSize(16, 16);
        dim3 numBlocks((res_x + blockSize.x - 1) / blockSize.x, 
                       (res_y + blockSize.y - 1) / blockSize.y);
        Camera* d_camera;
        AABB* d_bbox;
        hipMalloc(&d_camera, sizeof(Camera));
        hipMalloc(&d_bbox, sizeof(AABB));
        hipMemcpy(d_camera, &camera, sizeof(Camera), hipMemcpyHostToDevice);
        hipMemcpy(d_bbox, &bbox, sizeof(AABB), hipMemcpyHostToDevice);
        rayTracingKernel<<<numBlocks, blockSize>>>(frame_buffer, res_x, res_y, d_camera, d_bbox, volume, tf, 1.0f / sampling_rate);
        hipDeviceSynchronize();
        hipMemcpyToArray(array, 0, 0, frame_buffer, size, hipMemcpyDeviceToDevice);
        hipGraphicsUnmapResources(1, &cuda_resource);
        hipFree(d_camera);
        hipFree(d_bbox);
    }

    void render_host(const Camera& camera, const AABB& bbox, const VolumeData<float>* volume) {
        rayTracingHost(frame_buffer, res_x, res_y, &camera, &bbox, volume, tf, 1.0f / sampling_rate);
        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, res_x, res_y, 0, GL_RGBA, GL_UNSIGNED_BYTE, frame_buffer);
    }
};

GLFWwindow* createWindow(int width, int height, const char* title) {
    // Initialize GLFW
    if (!glfwInit()) {
        std::cerr << "Failed to initialize GLFW" << std::endl;
        return nullptr;
    }

    // Set OpenGL version and profile
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    // Create window
    GLFWwindow* window = glfwCreateWindow(width, height, "Hello GUI", nullptr, nullptr);
    if (!window) {
        std::cerr << "Failed to create GLFW window" << std::endl;
        glfwTerminate();
        return nullptr;
    }
    glfwMakeContextCurrent(window);
    glfwSwapInterval(0); // Disable VSync

    // Initialize GLEW
    glewExperimental = GL_TRUE;
    if (glewInit() != GLEW_OK) {
        std::cerr << "Failed to initialize GLEW" << std::endl;
        return nullptr;
    }
    return window;
} 

int main() {
    std::ifstream config_file("../config/config.json");
    json args = json::parse(config_file);
    uint32_t WIDTH = args["WIDTH"].get<uint32_t>();
    uint32_t HEIGHT = args["HEIGHT"].get<uint32_t>();
    uint32_t VIS_WIDTH = args["VIS_WIDTH"].get<uint32_t>();
    uint32_t CONFIG_WIDTH = WIDTH - VIS_WIDTH;
    float CAMERA_RADIUS = args["CAMERA_RADIUS"].get<float>();
    float CAMERA_SPEED = args["CAMERA_SPEED"].get<float>();
    bool USE_CPU = args["USE_CPU"].get<bool>();
    
    GLFWwindow* window = createWindow(WIDTH, HEIGHT, "Volume Renderer");
    if (window == nullptr) {
        return -1;
    }
    // Set background color
    glClearColor(0.1f, 0.1f, 0.1f, 1.0f);

    // Setup Dear ImGui context
    IMGUI_CHECKVERSION();
    ImGui::CreateContext();
    ImGuiIO& io = ImGui::GetIO(); (void)io;
    ImGui::StyleColorsDark();
    ImGui_ImplGlfw_InitForOpenGL(window, true);
    ImGui_ImplOpenGL3_Init("#version 130");

    VolumeRenderer renderer(VIS_WIDTH, HEIGHT, USE_CPU);
    renderer.init();

    const char* VIS_WINDOW_NAME = "Volume";
    ImVec2 VIS_WINDOW_POS = ImVec2(0, 0);
    ImVec2 VIS_WINDOW_SIZE = ImVec2(VIS_WIDTH, HEIGHT);
    const char* CONFIG_WINDOW_NAME = "Settings";
    ImVec2 CONFIG_WINDOW_POS = ImVec2(VIS_WIDTH, 0);
    ImVec2 CONFIG_WINDOW_SIZE = ImVec2(CONFIG_WIDTH, HEIGHT);

    // Camera
    glm::vec3 eye(-1.0f, 0.0f, 0.0f);
    glm::vec3 center(0.0f, 0.0f, 0.0f);
    glm::vec3 up = glm::vec3(args["CAMERA_UP"][0].get<float>(), args["CAMERA_UP"][1].get<float>(), args["CAMERA_UP"][2].get<float>());
    float phi = 0.0f;
    float theta = 0.0f;
    float radius = CAMERA_RADIUS;
    float camera_speed = CAMERA_SPEED;

    // load volume data
    glm::vec3 voxel_ratio(1.0f, 1.0f, 2.0f);
    DataLoader cthead_loader(std::string("../data/CThead.bin"), true);
    VolumeData<float>* cthead_h_volume = new VolumeData<float>(cthead_loader.getData(), cthead_loader.getSize(), voxel_ratio);
    VolumeData<float>* cthead_d_volume = cthead_h_volume->copyToDevice();
    DataLoader mrbrain_loader(std::string("../data/MRbrain.bin"), true);
    VolumeData<float>* mrbrain_h_volume = new VolumeData<float>(mrbrain_loader.getData(), mrbrain_loader.getSize(), voxel_ratio);
    VolumeData<float>* mrbrain_d_volume = mrbrain_h_volume->copyToDevice();
    
    VolumeData<float>* h_volume_array[] = {cthead_h_volume, mrbrain_h_volume};
    VolumeData<float>* d_volume_array[] = {cthead_d_volume, mrbrain_d_volume};
    int data_idx = 0;

    // set volume within [0, 1]^3
    glm::vec3 volume_shape = glm::vec3(cthead_h_volume->getShape()) * voxel_ratio;
    float max_dim = std::max(volume_shape.x, std::max(volume_shape.y, volume_shape.z));
    volume_shape /= max_dim;
    // move volume to the center
    AABB cthead_bbox(-volume_shape / 2.0f, volume_shape / 2.0f);

    volume_shape = glm::vec3(mrbrain_h_volume->getShape()) * voxel_ratio;
    max_dim = std::max(volume_shape.x, std::max(volume_shape.y, volume_shape.z));
    volume_shape /= max_dim;
    AABB mrbrain_bbox(-volume_shape / 2.0f, volume_shape / 2.0f);

    AABB* bbox_array[] = {&cthead_bbox, &mrbrain_bbox};

    const char* tf_names[] = {"Grayscale", "Skull", "Skin", "Nerve"};
    const char* data_names[] = {"CThead", "MRbrain"};

    // Main loop
    Timer timer;
    while (!glfwWindowShouldClose(window)) {
        glfwPollEvents();

        // Calculate delta time
        float deltaTime = timer.getDeltaTime();
        if (deltaTime > 0.033f) {
            // Update the camera
            theta = theta + camera_speed * deltaTime;
            theta = theta > 360.0f ? theta - 360.0f : theta;
            if (data_idx == 0) {
                eye = radius * glm::vec3(-cos(glm::radians(phi)) * cos(glm::radians(theta)),
                            cos(glm::radians(phi)) * sin(glm::radians(theta)),
                            sin(glm::radians(phi))
                            );
                up = glm::vec3(0.0f, 0.0f, 1.0f);
            } else {
                eye = radius * glm::vec3(cos(glm::radians(phi)) * cos(glm::radians(theta)),
                                sin(glm::radians(phi)),
                                cos(glm::radians(phi)) * sin(glm::radians(theta))
                                );
                up = glm::vec3(0.0f, 1.0f, 0.0f);
            }
            timer.update();
        }
        
        Camera camera(eye, center - eye, up, YAW, PITCH, FOV, float(VIS_WIDTH) / HEIGHT);
        if (renderer.on_host) {
            renderer.render_host(camera, *bbox_array[data_idx], h_volume_array[data_idx]);
        } else {
            renderer.render_device(camera, *bbox_array[data_idx], d_volume_array[data_idx]);
        }

        // Start the Dear ImGui frame
        ImGui_ImplOpenGL3_NewFrame();
        ImGui_ImplGlfw_NewFrame();
        ImGui::NewFrame();

        ImGui::SetWindowPos(VIS_WINDOW_NAME, VIS_WINDOW_POS);
        ImGui::SetWindowSize(VIS_WINDOW_NAME, VIS_WINDOW_SIZE);
        ImGui::Begin(VIS_WINDOW_NAME);
        ImGui::Image(renderer.gl_texture_id, VIS_WINDOW_SIZE);
        ImGui::End();

        ImGui::SetWindowPos(CONFIG_WINDOW_NAME, CONFIG_WINDOW_POS);
        ImGui::SetWindowSize(CONFIG_WINDOW_NAME, CONFIG_WINDOW_SIZE);
        ImGui::Begin(CONFIG_WINDOW_NAME);
        ImGui::Text("%.2f FPS", io.Framerate);
        ImGui::Combo("Data", &data_idx, data_names, IM_ARRAYSIZE(data_names));
        ImGui::Combo("Transfer Function", (int*)&renderer.tf, tf_names, IM_ARRAYSIZE(tf_names));
        ImGui::SliderFloat("Sampling Rate", &renderer.sampling_rate, 1.0f, 500.0f);
        ImGui::End();

        ImGui::Render();

        glClear(GL_COLOR_BUFFER_BIT);
        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());
        
        // Swap buffers
        glfwSwapBuffers(window);
    }

    // Clean up
    cthead_h_volume->release();
    hipFree(cthead_d_volume);
    mrbrain_h_volume->release();
    hipFree(mrbrain_d_volume);

    ImGui_ImplOpenGL3_Shutdown();
    ImGui_ImplGlfw_Shutdown();
    ImGui::DestroyContext();

    glfwDestroyWindow(window);
    glfwTerminate();
    return 0;
}
