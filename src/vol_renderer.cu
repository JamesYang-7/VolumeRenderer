#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <cuda_gl_interop.h>
#include <imgui.h>
#include <imgui_impl_glfw.h>
#include <imgui_impl_opengl3.h>
#include <tiny-cuda-nn/common_host.h>
#include "vol_renderer/common.h"
#include "vol_renderer/camera.h"
#include "vol_renderer/ray.h"
#include "vol_renderer/bbox.h"
#include "vol_renderer/volume_data.h"
#include "vol_renderer/data_loader.h"
#include "vol_renderer/transfer_function.h"
#include "vol_renderer/timer.h"


__global__ void processTextureKernel(uchar4* frame_buffer, uint32_t width, uint32_t height) {
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    uint32_t idx = y * width + x;

    frame_buffer[idx].x = (uint32_t)((float)x / width * 255.0f);
    frame_buffer[idx].y = (uint32_t)((float)y / height * 255.0f);
    frame_buffer[idx].z = 127;
}

__host__ __device__ void composite(
    const glm::vec3& color,
    const float& alpha,
    const glm::vec3& bg_color,
    const float& bg_alpha,
    glm::vec3& out_color,
    float& out_alpha
)
{
    out_color = alpha * color + (1.0f - alpha) * bg_color;
    out_alpha = alpha + (1.0f - alpha) * bg_alpha;
}

__global__ void rayTracingKernel(
    uchar4* frame_buffer, uint32_t width,
    uint32_t height, const Camera* camera,
    const AABB* bbox,
    const VolumeData<float>* volume,
    TransferFunctionType tf_type,
    const float step_size = 0.01f
)
{
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    uint32_t idx = y * width + x;

    float u = ((float)x + 0.5f) / width;
    float v = ((float)y + 0.5f) / height;

    Ray ray = camera->generateRay(u, v);
    float tmin, tmax;
    
    glm::vec3 color(0.0f, 0.0f, 0.5f);
    float alpha = 0.2f;
    glm::vec3 bg_color(0.0f);
    float bg_alpha = 0.0f;
    glm::vec3 out_color(0.0f);
    float out_alpha = 0.0f;
    unsigned char r = 0, g = 0, b = 0;
    if (bbox->ray_intersect(&ray, &tmin, &tmax)) {
        float t = tmax;
        while (t > tmin) {
            glm::vec3 p = ray.at(t);
            glm::vec3 local_pos = bbox->getLocalPos(p);
            float val = volume->at(local_pos);
            glm::vec4 rgba = getColor(val, tf_type);
            color = glm::vec3(rgba.x, rgba.y, rgba.z);
            alpha = rgba.w;
            composite(color, alpha, bg_color, bg_alpha, out_color, out_alpha);
            bg_color = out_color;
            bg_alpha = out_alpha;
            t -= step_size;
        }
        r = (unsigned char)(255.0f * out_color.x);
        g = (unsigned char)(255.0f * out_color.y);
        b = (unsigned char)(255.0f * out_color.z);
    }
    frame_buffer[idx].x = r;
    frame_buffer[idx].y = g;
    frame_buffer[idx].z = b;
}

void rayTracingHost(
    uchar4* frame_buffer, uint32_t width,
    uint32_t height, const Camera* camera,
    const AABB* bbox,
    const VolumeData<float>* volume,
    TransferFunctionType tf_type,
    const float step_size = 0.01f
)
{
    for (uint32_t x = 0; x < width; ++x) {
        for (uint32_t y = 0; y < height; ++y) {
            uint32_t idx = y * width + x;
            float u = ((float)x + 0.5f) / width;
            float v = ((float)y + 0.5f) / height;

            Ray ray = camera->generateRay(u, v);
            float tmin, tmax;
            glm::vec3 color(0.0f, 0.0f, 0.5f);
            float alpha = 0.2f;
            glm::vec3 bg_color(0.0f);
            float bg_alpha = 0.0f;
            glm::vec3 out_color(0.0f);
            float out_alpha = 0.0f;
            unsigned char r = 0, g = 0, b = 0;
            if (bbox->ray_intersect(&ray, &tmin, &tmax)) {
                float t = tmax;
                while (t > tmin) {
                    glm::vec3 p = ray.at(t);
                    glm::vec3 local_pos = bbox->getLocalPos(p);
                    float val = volume->at(local_pos);
                    glm::vec4 rgba = getColor(val, tf_type);
                    color = glm::vec3(rgba.x, rgba.y, rgba.z);
                    alpha = rgba.w;
                    composite(color, alpha, bg_color, bg_alpha, out_color, out_alpha);
                    bg_color = out_color;
                    bg_alpha = out_alpha;
                    t -= step_size;
                }
                r = (unsigned char)(255.0f * out_color.x);
                g = (unsigned char)(255.0f * out_color.y);
                b = (unsigned char)(255.0f * out_color.z);
            }
            frame_buffer[idx].x = r;
            frame_buffer[idx].y = g;
            frame_buffer[idx].z = b;
        }
    }
}


struct VolumeRenderer {
    uint32_t res_x = 0;
    uint32_t res_y = 0;
    GLuint gl_texture_id;
    hipGraphicsResource* cuda_resource;
    uchar4* frame_buffer;
    size_t size;
    TransferFunctionType tf = TransferFunctionType::GRAYSCALE;
    float sampling_rate = 10.0f; // inverse of step size
    bool on_host = false;

    VolumeRenderer(uint32_t res_x, uint32_t res_y, bool on_host=false) : res_x(res_x), res_y(res_y), on_host(on_host) {
        size = res_x * res_y * sizeof(uchar4);
        if (on_host) {
            frame_buffer = (uchar4*)malloc(size);
            memset(frame_buffer, 255, size);
        } else {
            hipMalloc(&frame_buffer, size);
            hipMemset(frame_buffer, 255, size);
        }
    }

    ~VolumeRenderer() {
        if (on_host) {
            free(frame_buffer);
        } else {
            hipGraphicsUnregisterResource(cuda_resource);
            hipFree(frame_buffer);
        }
    }

    void init() {
        glGenTextures(1, &gl_texture_id);
        glBindTexture(GL_TEXTURE_2D, gl_texture_id);
        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, res_x, res_y, 0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
        if (!on_host) hipGraphicsGLRegisterImage(&cuda_resource, gl_texture_id, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
    }

    void renderTexture() {
        hipArray* array;
        hipGraphicsMapResources(1, &cuda_resource);
        hipGraphicsSubResourceGetMappedArray(&array, cuda_resource, 0, 0);

        dim3 blockSize(16, 16);
        dim3 numBlocks((res_x + blockSize.x - 1) / blockSize.x, 
                       (res_y + blockSize.y - 1) / blockSize.y);
        processTextureKernel<<<numBlocks, blockSize>>>(frame_buffer, res_x, res_y);

        hipMemcpyToArray(array, 0, 0, frame_buffer, size, hipMemcpyDeviceToDevice);
        hipDeviceSynchronize();
        hipGraphicsUnmapResources(1, &cuda_resource);
    }

    void render(const Camera& camera, const AABB& bbox, const VolumeData<float>* d_volume) {
        if (on_host) {
            render_host(camera, bbox, d_volume);
        } else {
            render_device(camera, bbox, d_volume);
        }
    }

    void render_device(const Camera& camera, const AABB& bbox, const VolumeData<float>* d_volume) {
        hipArray* array;
        hipGraphicsMapResources(1, &cuda_resource);
        hipGraphicsSubResourceGetMappedArray(&array, cuda_resource, 0, 0);

        dim3 blockSize(16, 16);
        dim3 numBlocks((res_x + blockSize.x - 1) / blockSize.x, 
                       (res_y + blockSize.y - 1) / blockSize.y);
        Camera* d_camera;
        AABB* d_bbox;
        hipMalloc(&d_camera, sizeof(Camera));
        hipMalloc(&d_bbox, sizeof(AABB));
        hipMemcpy(d_camera, &camera, sizeof(Camera), hipMemcpyHostToDevice);
        hipMemcpy(d_bbox, &bbox, sizeof(AABB), hipMemcpyHostToDevice);
        rayTracingKernel<<<numBlocks, blockSize>>>(frame_buffer, res_x, res_y, d_camera, d_bbox, d_volume, tf, 1.0f / sampling_rate);
        hipDeviceSynchronize();
        hipMemcpyToArray(array, 0, 0, frame_buffer, size, hipMemcpyDeviceToDevice);
        hipGraphicsUnmapResources(1, &cuda_resource);
        hipFree(d_camera);
        hipFree(d_bbox);
    }

    void render_host(const Camera& camera, const AABB& bbox, const VolumeData<float>* d_volume) {
        rayTracingHost(frame_buffer, res_x, res_y, &camera, &bbox, d_volume, tf, 1.0f / sampling_rate);
        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, res_x, res_y, 0, GL_RGBA, GL_UNSIGNED_BYTE, frame_buffer);
    }
};

GLFWwindow* createWindow(int width, int height, const char* title) {
    // Initialize GLFW
    if (!glfwInit()) {
        std::cerr << "Failed to initialize GLFW" << std::endl;
        return nullptr;
    }

    // Set OpenGL version and profile
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    // Create window
    GLFWwindow* window = glfwCreateWindow(WIDTH, HEIGHT, "Hello GUI", nullptr, nullptr);
    if (!window) {
        std::cerr << "Failed to create GLFW window" << std::endl;
        glfwTerminate();
        return nullptr;
    }
    glfwMakeContextCurrent(window);
    glfwSwapInterval(0); // Disable VSync

    // Initialize GLEW
    glewExperimental = GL_TRUE;
    if (glewInit() != GLEW_OK) {
        std::cerr << "Failed to initialize GLEW" << std::endl;
        return nullptr;
    }
    return window;
} 

int main() {
    
    GLFWwindow* window = createWindow(WIDTH, HEIGHT, "Volume Renderer");
    if (window == nullptr) {
        return -1;
    }
    // Set background color
    glClearColor(0.1f, 0.1f, 0.1f, 1.0f);

    // Setup Dear ImGui context
    IMGUI_CHECKVERSION();
    ImGui::CreateContext();
    ImGuiIO& io = ImGui::GetIO(); (void)io;
    ImGui::StyleColorsDark();
    ImGui_ImplGlfw_InitForOpenGL(window, true);
    ImGui_ImplOpenGL3_Init("#version 130");

    VolumeRenderer renderer(VIS_WIDTH, HEIGHT, USE_CPU);
    renderer.init();

    const char* VIS_WINDOW_NAME = "Volume";
    ImVec2 VIS_WINDOW_POS = ImVec2(0, 0);
    ImVec2 VIS_WINDOW_SIZE = ImVec2(VIS_WIDTH, HEIGHT);
    const char* CONFIG_WINDOW_NAME = "Settings";
    ImVec2 CONFIG_WINDOW_POS = ImVec2(VIS_WIDTH, 0);
    ImVec2 CONFIG_WINDOW_SIZE = ImVec2(CONFIG_WIDTH, HEIGHT);

    // Camera
    glm::vec3 eye(-1.0f, 0.0f, 0.0f);
    glm::vec3 center(0.0f, 0.0f, 0.0f);
    glm::vec3 up(0.0f, 0.0f, 1.0f);
    float phi = 0.0f;
    float theta = 0.0f;
    float radius = CAMERA_RADIUS;
    float camera_speed = CAMERA_SPEED;

    // load volume data
    DataLoader loader("../data/CThead.bin", true);
    glm::vec3 voxel_ratio(1.0f, 1.0f, 2.0f);
    VolumeData<float> volume(loader.getData(), loader.getSize(), voxel_ratio);
    const float* h_volume_data = loader.getData();
    Voxel* h_voxels = volume.getVoxels();
    VolumeData<float> h_volume(h_volume_data, h_voxels, loader.getSize(), voxel_ratio);
    float* d_volume_data;
    Voxel* d_voxels;
    hipMalloc(&d_volume_data, loader.getNum() * sizeof(float));
    hipMalloc(&d_voxels, volume.getNum() * sizeof(Voxel));
    hipMemcpy(d_volume_data, h_volume_data, loader.getNum() * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_voxels, h_voxels, volume.getNum() * sizeof(Voxel), hipMemcpyHostToDevice);
    VolumeData<float>* d_volume;
    volume.setData(d_volume_data);
    volume.setVoxels(d_voxels);
    hipMalloc(&d_volume, sizeof(VolumeData<float>));
    hipMemcpy(d_volume, &volume, sizeof(VolumeData<float>), hipMemcpyHostToDevice);

    // set volume within [0, 1]^3
    glm::vec3 volume_shape = glm::vec3(volume.getShape()) * voxel_ratio;
    float max_dim = std::max(volume_shape.x, std::max(volume_shape.y, volume_shape.z));
    volume_shape /= max_dim;
    // move volume to the center
    AABB bbox(-volume_shape / 2.0f, volume_shape / 2.0f);

    const char* tf_names[] = {"GRAYSCALE", "TF1", "TF2"};

    // Main loop
    Timer timer;
    while (!glfwWindowShouldClose(window)) {
        glfwPollEvents();

        // Calculate delta time
        float deltaTime = timer.getDeltaTime();
        if (deltaTime > 0.033f) {
            // Update the camera
            theta = theta + camera_speed * deltaTime;
            theta = theta > 360.0f ? theta - 360.0f : theta;
            eye = radius * glm::vec3(-cos(glm::radians(phi)) * cos(glm::radians(theta)),
                            cos(glm::radians(phi)) * sin(glm::radians(theta)),
                            sin(glm::radians(phi))
                            );
            timer.update();
        }
        
        Camera camera(eye, center - eye, up, YAW, PITCH, FOV, float(VIS_WIDTH) / HEIGHT);
        if (renderer.on_host) {
            renderer.render_host(camera, bbox, &h_volume);
        } else {
            renderer.render_device(camera, bbox, d_volume);
        }

        // Start the Dear ImGui frame
        ImGui_ImplOpenGL3_NewFrame();
        ImGui_ImplGlfw_NewFrame();
        ImGui::NewFrame();

        ImGui::SetWindowPos(VIS_WINDOW_NAME, VIS_WINDOW_POS);
        ImGui::SetWindowSize(VIS_WINDOW_NAME, VIS_WINDOW_SIZE);
        ImGui::Begin(VIS_WINDOW_NAME);
        ImGui::Image(renderer.gl_texture_id, VIS_WINDOW_SIZE);
        ImGui::End();

        ImGui::SetWindowPos(CONFIG_WINDOW_NAME, CONFIG_WINDOW_POS);
        ImGui::SetWindowSize(CONFIG_WINDOW_NAME, CONFIG_WINDOW_SIZE);
        ImGui::Begin(CONFIG_WINDOW_NAME);
        ImGui::Text("%.2f FPS", io.Framerate);
        ImGui::Combo("Transfer Function", (int*)&renderer.tf, tf_names, IM_ARRAYSIZE(tf_names));
        ImGui::SliderFloat("Sampling Rate", &renderer.sampling_rate, 1.0f, 200.0f);
        ImGui::End();

        ImGui::Render();

        glClear(GL_COLOR_BUFFER_BIT);
        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());
        
        // Swap buffers
        glfwSwapBuffers(window);
    }

    // Clean up

    hipFree(d_volume_data);
    hipFree(d_voxels);
    hipFree(d_volume);
    free(h_voxels);

    ImGui_ImplOpenGL3_Shutdown();
    ImGui_ImplGlfw_Shutdown();
    ImGui::DestroyContext();

    glfwDestroyWindow(window);
    glfwTerminate();
    return 0;
}
