#include "hip/hip_runtime.h"
#include <GL/glew.h>
#include <GLFW/glfw3.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <cuda_gl_interop.h>
#include <imgui.h>
#include <imgui_impl_glfw.h>
#include <imgui_impl_opengl3.h>
#include "vol_renderer/common.h"
#include "vol_renderer/camera.h"
#include "vol_renderer/ray.h"
#include "vol_renderer/bbox.h"
#include <tiny-cuda-nn/common_host.h>


__global__ void processTextureKernel(uchar4* frame_buffer, uint32_t width, uint32_t height) {
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    uint32_t idx = y * width + x;

    frame_buffer[idx].x = (uint32_t)((float)x / width * 255.0f);
    frame_buffer[idx].y = (uint32_t)((float)y / height * 255.0f);
    frame_buffer[idx].z = 127;
}

__device__ void composite(
    const glm::vec3& color,
    const float& alpha,
    const glm::vec3& bg_color,
    const float& bg_alpha,
    glm::vec3& out_color,
    float& out_alpha
)
{
    out_color = alpha * color + (1.0f - alpha) * bg_color;
    out_alpha = alpha + (1.0f - alpha) * bg_alpha;
}

__global__ void rayTracingKernel(uchar4* frame_buffer, uint32_t width, uint32_t height, const Camera* camera, const AABB* bbox) {
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    if (x >= width || y >= height) return;
    uint32_t idx = y * width + x;

    float u = ((float)x + 0.5f) / width;
    float v = ((float)y + 0.5f) / height;

    Ray ray = camera->generateRay(u, v);
    float tmin, tmax;
    float step_size = 0.1f;
    
    glm::vec3 color(0.0f, 0.0f, 0.5f);
    float alpha = 0.2f;
    glm::vec3 bg_color(0.0f);
    float bg_alpha = 0.0f;
    glm::vec3 out_color(0.0f);
    float out_alpha = 0.0f;
    unsigned char r = 0, g = 0, b = 0;
    if (bbox->ray_intersect(&ray, &tmin, &tmax)) {
        float t = tmax;
        while (t > tmin) {
            glm::vec3 p = ray.at(t);
            composite(color, alpha, bg_color, bg_alpha, out_color, out_alpha);
            bg_color = out_color;
            bg_alpha = out_alpha;
            t -= step_size;
        }
        r = (unsigned char)(255.0f * out_color.x);
        g = (unsigned char)(255.0f * out_color.y);
        b = (unsigned char)(255.0f * out_color.z);
    }
    frame_buffer[idx].x = r;
    frame_buffer[idx].y = g;
    frame_buffer[idx].z = b;
}


struct VolumeRenderer {
    uint32_t res_x = 0;
    uint32_t res_y = 0;
    GLuint gl_texture_id;
    hipGraphicsResource* cuda_resource;
    hipSurfaceObject_t surface_obj;
    uchar4* frame_buffer;
    size_t size;

    VolumeRenderer(uint32_t res_x, uint32_t res_y) : res_x(res_x), res_y(res_y) {
        size = res_x * res_y * sizeof(uchar4);
        hipMalloc(&frame_buffer, size);
        hipMemset(frame_buffer, 255, size);
    }

    ~VolumeRenderer() {
        hipGraphicsUnregisterResource(cuda_resource);
        hipFree(frame_buffer);
    }

    void init() {
        glGenTextures(1, &gl_texture_id);
        glBindTexture(GL_TEXTURE_2D, gl_texture_id);
        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA, res_x, res_y, 0, GL_RGBA, GL_UNSIGNED_BYTE, nullptr);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
        glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
        hipGraphicsGLRegisterImage(&cuda_resource, gl_texture_id, GL_TEXTURE_2D, hipGraphicsRegisterFlagsSurfaceLoadStore);
    }

    void renderTexture() {
        hipArray* array;
        hipGraphicsMapResources(1, &cuda_resource);
        hipGraphicsSubResourceGetMappedArray(&array, cuda_resource, 0, 0);

        dim3 blockSize(16, 16);
        dim3 numBlocks((res_x + blockSize.x - 1) / blockSize.x, 
                       (res_y + blockSize.y - 1) / blockSize.y);
        processTextureKernel<<<numBlocks, blockSize>>>(frame_buffer, res_x, res_y);

        hipMemcpyToArray(array, 0, 0, frame_buffer, size, hipMemcpyDeviceToDevice);
        hipDeviceSynchronize();
        hipGraphicsUnmapResources(1, &cuda_resource);
    }

    void render(const Camera& camera, const AABB& bbox) {
        hipArray* array;
        hipGraphicsMapResources(1, &cuda_resource);
        hipGraphicsSubResourceGetMappedArray(&array, cuda_resource, 0, 0);

        dim3 blockSize(16, 16);
        dim3 numBlocks((res_x + blockSize.x - 1) / blockSize.x, 
                       (res_y + blockSize.y - 1) / blockSize.y);
        Camera* d_camera;
        AABB* d_bbox;
        hipMalloc(&d_camera, sizeof(Camera));
        hipMalloc(&d_bbox, sizeof(AABB));
        hipMemcpy(d_camera, &camera, sizeof(Camera), hipMemcpyHostToDevice);
        hipMemcpy(d_bbox, &bbox, sizeof(AABB), hipMemcpyHostToDevice);
        rayTracingKernel<<<numBlocks, blockSize>>>(frame_buffer, res_x, res_y, d_camera, d_bbox);
        hipDeviceSynchronize();
        hipMemcpyToArray(array, 0, 0, frame_buffer, size, hipMemcpyDeviceToDevice);
        hipGraphicsUnmapResources(1, &cuda_resource);
        hipFree(d_camera);
        hipFree(d_bbox);
    }
};

GLFWwindow* createWindow(int width, int height, const char* title) {
    // Initialize GLFW
    if (!glfwInit()) {
        std::cerr << "Failed to initialize GLFW" << std::endl;
        return nullptr;
    }

    // Set OpenGL version and profile
    glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
    glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
    glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);

    // Create window
    GLFWwindow* window = glfwCreateWindow(WIDTH, HEIGHT, "Hello GUI", nullptr, nullptr);
    if (!window) {
        std::cerr << "Failed to create GLFW window" << std::endl;
        glfwTerminate();
        return nullptr;
    }
    glfwMakeContextCurrent(window);

    // Initialize GLEW
    glewExperimental = GL_TRUE;
    if (glewInit() != GLEW_OK) {
        std::cerr << "Failed to initialize GLEW" << std::endl;
        return nullptr;
    }
    return window;
} 

int main() {
    
    GLFWwindow*  window = createWindow(WIDTH, HEIGHT, "Volume Renderer");
    if (window == nullptr) {
        return -1;
    }
    // Set background color
    glClearColor(0.1f, 0.1f, 0.1f, 1.0f);

    // Setup Dear ImGui context
    IMGUI_CHECKVERSION();
    ImGui::CreateContext();
    ImGuiIO& io = ImGui::GetIO(); (void)io;
    ImGui::StyleColorsDark();
    ImGui_ImplGlfw_InitForOpenGL(window, true);
    ImGui_ImplOpenGL3_Init("#version 130");

    VolumeRenderer renderer(WIDTH, HEIGHT);
    renderer.init();

    const char* VIS_WINDOW_NAME = "Volume";
    ImVec2 VIS_WINDOW_POS = ImVec2(0, 0);
    ImVec2 VIS_WINDOW_SIZE = ImVec2(WIDTH, HEIGHT);

    // scene
    AABB bbox(glm::vec3(-1.0f), glm::vec3(1.0f));
    glm::vec3 eye(2.0f, 2.0f, -2.0f);
    glm::vec3 center(0.0f, 0.0f, 0.0f);
    glm::vec3 up(0.0f, 0.0f, 1.0f);
    Camera camera(eye, center - eye, up);

    // Main loop
    float currentTime = 0.0f;
    float deltaTime = 0.0f;
    float lastTime = 0.0f;
    while (!glfwWindowShouldClose(window)) {
        glfwPollEvents();

        // Calculate delta time
        currentTime = glfwGetTime();
        deltaTime = currentTime - lastTime;
        if (deltaTime < 0.5f) continue;
        lastTime = currentTime;

        renderer.render(camera, bbox);

        // Start the Dear ImGui frame
        ImGui_ImplOpenGL3_NewFrame();
        ImGui_ImplGlfw_NewFrame();
        ImGui::NewFrame();

        // glBindTexture(GL_TEXTURE_2D, texture);
        ImGui::SetWindowPos(VIS_WINDOW_NAME, VIS_WINDOW_POS);
        ImGui::SetWindowSize(VIS_WINDOW_NAME, VIS_WINDOW_SIZE);
        ImGui::Begin(VIS_WINDOW_NAME);
        ImGui::Image(renderer.gl_texture_id, ImVec2(WIDTH, HEIGHT));
        ImGui::End();
        ImGui::Render();

        glClear(GL_COLOR_BUFFER_BIT);
        ImGui_ImplOpenGL3_RenderDrawData(ImGui::GetDrawData());
        
        // Swap buffers
        glfwSwapBuffers(window);
    }

    // Clean up
    ImGui_ImplOpenGL3_Shutdown();
    ImGui_ImplGlfw_Shutdown();
    ImGui::DestroyContext();

    glfwDestroyWindow(window);
    glfwTerminate();
    return 0;
}
